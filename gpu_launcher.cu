#include "gpu_launcher.h"
#include "gpu_integral.h"
#include <hip/hip_runtime.h>
#include <iostream>

void launchFloatKernel(int* h_n_vals, float* h_m_vals, float* h_results, int total) {
    int* d_n_vals;
    float* d_m_vals;
    float* d_results;

    hipMalloc(&d_n_vals, sizeof(int) * total);
    hipMalloc(&d_m_vals, sizeof(float) * total);
    hipMalloc(&d_results, sizeof(float) * total);

    hipMemcpy(d_n_vals, h_n_vals, sizeof(int) * total, hipMemcpyHostToDevice);
    hipMemcpy(d_m_vals, h_m_vals, sizeof(float) * total, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (total + blockSize - 1) / blockSize;

    computeExpIntegralKernel<<<gridSize, blockSize>>>(d_results, d_n_vals, d_m_vals, total);

    hipMemcpy(h_results, d_results, sizeof(float) * total, hipMemcpyDeviceToHost);

    hipFree(d_n_vals);
    hipFree(d_m_vals);
    hipFree(d_results);
}

